﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <random>

using namespace std;

__device__ void swap_rows(double* device_matrix_data, int N, int index, int k, int size) {
    int t_index = threadIdx.x;
    double tmp;
    for (int i = t_index; i < N + 1; i += size) {
        tmp = device_matrix_data[index + i];
        device_matrix_data[index + i] = device_matrix_data[k * (N + 1) + i];
        device_matrix_data[k * (N + 1) + i] = tmp;
    }
}

__global__ void find_pivot_row(double* device_matrix_data, int N, int k, int N_rows_per_thread, int size) {
    // Searching for the row with maximum nonzero first element and moving this row to the top
    extern __shared__ double s[];
    double* max_elems = s;
    int* indexes = (int*)&s[size];

    int t_index = threadIdx.x;
    int row_start = (k + N_rows_per_thread * t_index) * (N + 1);
    int row_stop = (k + N_rows_per_thread * (t_index + 1)) * (N + 1);
    if (row_start > (N - 1) * (N + 1))
        return;
    if (row_stop > N * (N + 1))
        row_stop = N * (N + 1);

    double max_elem = 0;
    int index = 0;
    for (int i = row_start; i < row_stop; i += N + 1) {
        double first_elem = abs(device_matrix_data[i + k]);
        if (max_elem < first_elem) {
            max_elem = first_elem;
            index = i;
        }
    }

    max_elems[t_index] = max_elem;
    indexes[t_index] = index;
    __syncthreads();
    
    for (int i = size / 2; i > 0; i /= 2) {
        if (t_index < i) {
            if (max_elems[i] < max_elems[t_index + i]) {
                max_elems[i] = max_elems[t_index + i];
                indexes[i] = indexes[t_index + i];
            }
        }
        __syncthreads();
    }

    double a = device_matrix_data[indexes[0] + k];
    for (int i = t_index; i < N + 1; i += size)
        device_matrix_data[indexes[0] + k + i] /= a;
    swap_rows(device_matrix_data, N, indexes[0], k, size);
}

__global__ void subtract_rows(double* device_matrix_data, int N, int k, int size) {
    // Each thread calculates one row at a time
    int t_index = threadIdx.x + blockDim.x * blockIdx.x;
    if (t_index > N) return;

    for (int i = k + 1 + t_index; i < N; i += size) {
        //normalize_row(device_matrix_data, N, i * (N + 1), k, 1);
        double a = device_matrix_data[i * (N + 1) + k];
        for (int j = k; j < N + 1; j++) {
            device_matrix_data[i * (N + 1) + j] /= a;
            device_matrix_data[i * (N + 1) + j] -= device_matrix_data[k * (N + 1) + j];
        }
    }
}

__global__ void init_X(double* device_matrix_data, double* device_X, int N, int size) {
    int t_index = threadIdx.x + blockDim.x * blockIdx.x;
    if (t_index > N) return;

    for (int i = t_index; i < N; i += size)
        device_X[i] = device_matrix_data[i * (N + 1) + N];
}

__global__ void calculate_X(double* device_matrix_data, double* device_X, int N, int k, int size) {
    int t_index = threadIdx.x + blockDim.x * blockIdx.x;
    if (t_index > N) return;
    
    for (int i = t_index; i < k; i += size)
        device_X[i] -= device_matrix_data[i * (N + 1) + k] * device_X[k];
}

void print_matrix(double** matrix, int N_rows, int N_cols = -1, int limit_cols = 10, int limit_rows = 10) {
    if (N_cols == -1) N_cols = N_rows;
    for (int i = 0; i < N_rows; i++) {
        if (i != limit_rows) {
            cout << "|\t";
            for (int j = 0; j < N_cols + 1; j++) {
                if (j != limit_cols)
                    cout << setprecision(3) << matrix[i][j] << "\t";
                else {
                    cout << "\t...";
                    break;
                }
            }
            cout << "\t|\n";
        }
        else {
            cout << "|\t......\t";
            for (int j = 1; j < limit_cols + 1; j++)
                cout << "......\t";
            cout << "\t|\n";
            break;
        }
    }
    cout << endl;
}

void print_X(double* X, const int& N, int limit_cols = 8) {
    for (int i = 0; i < N - 1; i++) {
        if (i != limit_cols)
            cout << "X" << i + 1 << " = " << X[i] << ", ";
        else {
            cout << "..., ";
            break;
        }
    }
    cout << "X" << N << " = " << X[N - 1] << ".\n";
}

void print_time(double const& time_taken) {
    cout << "Time taken: " << setprecision(5) << 1e-3 * time_taken << " seconds.\n";
}

int main()
{
    int N;
    double min = -100, max = 100;
    double** matrix, * matrix_data, ** device_matrix, * device_matrix_data;
    float time_taken;
    int size;

    cout << "Select number of equations: ";
    cin >> N;

    // Generating random double in range
    random_device rd; // obtain a random number from hardware
    mt19937 gen(0);//rd()); // seed the generator
    uniform_real_distribution<> distr(min, max); // define the range

    // Filling the matrix with random float numbers
    matrix_data = new double[N * (N + 1)];
    matrix = new double* [N];
    for (int i = 0; i < N; i++)
        matrix[i] = &(matrix_data[i * (N + 1)]);

    for (int i = 0; i < N; i++)
        for (int j = 0; j < N + 1; j++)
            matrix[i][j] = distr(gen);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Timer start
    hipEventRecord(start, 0);

    hipMalloc((void**)&device_matrix_data, N * (N + 1) * sizeof(double));
    hipMemcpy(device_matrix_data, matrix_data, N * (N + 1) * sizeof(double), hipMemcpyHostToDevice);

    dim3 N_threads(128);
    int N_blocks;

    // Forward elimination
    for (int k = 0; k < N; k++) {
        const int N_rows_per_thread = 16;
        int N_threads_per_row = (N - k) / N_rows_per_thread;
        if ((N - k) % N_rows_per_thread != 0) N_threads_per_row++;
        int smem_size = N_threads_per_row * sizeof(double) + N_threads_per_row * sizeof(int);
        N_blocks = 1;
        int size = N_threads_per_row;


        find_pivot_row<<< N_blocks, N_threads_per_row, smem_size >>>(device_matrix_data, N, k, N_rows_per_thread, size);

        N_blocks = (N - k) / N_threads.x;
        if ((N - k) % N_threads.x != 0) N_blocks++;
        size = N_blocks * N_threads.x;

        subtract_rows<<< N_blocks, N_threads.x >>>(device_matrix_data, N, k, size);

    }

    // Back substitution
    double* X = new double[N];
    double* device_X;
    hipMalloc((void**)&device_X, N * sizeof(double));

    N_blocks = N / N_threads.x;
    if (N % N_threads.x != 0) N_blocks++;
    size = N_blocks * N_threads.x;
    init_X <<< N_blocks, N_threads.x >>> (device_matrix_data, device_X, N, size);

    for (int k = N - 1; k >= 0; k--) {
        N_blocks = k / N_threads.x;
        if (k % N_threads.x != 0) N_blocks++;
        size = N_blocks * N_threads.x;
        calculate_X<<< N_blocks, N_threads.x >>>(device_matrix_data, device_X, N, k, size);
    }

    hipMemcpy(X, device_X, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(matrix_data, device_matrix_data, N * (N + 1) * sizeof(double), hipMemcpyDeviceToHost);

    // Timer end
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_taken, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    // Showing results
    print_matrix(matrix, N);
    print_X(X, N);
    print_time(time_taken);

    delete[] X;
    hipFree(device_X);
    hipFree(device_matrix_data);
    delete[] matrix_data;
    delete[] matrix;
    return 0;
}
